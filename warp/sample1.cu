
#include <hip/hip_runtime.h>
#include <stdio.h>

int main (int argc, char** argv)
{
	int count;
	hipGetDeviceCount(&count);

	for (int i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);

		printf ("device #%d/%d\n", i+1, count);
 		printf ("    name: %s\n", prop.name);
		printf ("    warpSize: %d\n", prop.warpSize);
	}

	return 0;	
}

