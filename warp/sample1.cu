#include <stdio.h>
#include "../common/common.h"

int main (int argc, char** argv)
{
	int count;
	CHECK(hipGetDeviceCount(&count));

	for (int i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		CHECK(hipGetDeviceProperties(&prop, i));

		printf ("device #%d/%d\n", i+1, count);
 		printf ("    name: %s\n", prop.name);
		printf ("    warpSize: %d\n", prop.warpSize);
	}

	return 0;	
}

